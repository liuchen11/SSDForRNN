#include "hip/hip_runtime.h"
#include <math.h>

void sigmoid_unit(float v){
    return 1.0/(1.0+exp(-v));
}

void dsigmoid_unit(float v){
    return sigmoid_unit(v)*(1.0-sigmoid_unit(v));
}

// function sigmoid
// 1 block, 1 dimensional block size
__global__ void sigmoid(float* m1, float* m2, int n){
    for(int index=threadIdx.x;index<n;index+=blockDim.x)
        m2[index]=sigmoid_unit(m1[index]);
}

// function sigmoid
// 1 block, 1 dimensional block size
__global__ void dsigmoid(float* m1,float* m2, int n){
    for(int index=threadIdx.x;index<n;index+=blockDim.x)
        m2[index]=dsigmoid_unit(m1[index]);
}