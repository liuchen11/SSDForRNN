
#include <hip/hip_runtime.h>
// function: oneXn
// m1 of shape H*L, m2 of shape N*L*M, m3 of shape N*H*M
// N blocks, 2 dimensional block size
__global__ void oneXn(float* m1, float* m2, float* m3,
    int size_n, int size_l, int size_h, int size_m){
    int n=size_n,l=size_l, h=size_h, m=size_m;

    for(int h_index=threadIdx.x;h_index<h;h_index+=blockDim.x){
        for(int m_index=threadIdx.y;m_index<m;m_index+=blockDim.y){
            float value=0.0;
            for(int l_index=0;l_index<l;l_index++){
                float x=m1[h_index*l+l_index];
                float y=m2[blockIdx.x*l*m+l_index*m+m_index];
                value+=x*y;
            }
            m3[blockIdx.x*h*m+h_index*m+m_index]=value;
        }
    }
}

// function: nXone
// m1 of shape N*H*L, m2 of shape L*M, m3 of shape N*H*M
// N blocks, 2 dimensional block size
__global__ void nXone(float* m1, float* m2, float* m3,
    int size_n, int size_l, int size_h, int size_m){
    int n=size_n, l=size_l, h=size_h, m=size_m;

    for(int h_index=threadIdx.x;h_index<h;h_index+=blockDim.x){
        for(int m_index=threadIdx.y;m_index<m;m_index+=blockDim.y){
            float value=0.0;
            for(int l_index=0;l_index<l;l_index++){
                float x=m1[blockIdx.x*h*l+h_index*l+l_index];
                float y=m2[l_index*m+m_index];
                value+=x*y;
            }
            m3[blockIdx.x*h*m+h_index*m+m_index]=value;
        }
    }
}