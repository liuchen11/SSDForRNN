
#include <hip/hip_runtime.h>
#include <math.h>

// function add
// m1, m2 are vectors, matrices or tensors of the same size
// 1 block, 1 dimensional block size
__global__ void add(float* m1, float* m2, float* m3, int n){
    for(int index=threadIdx.x;index<n;index+=blockDim.x)
        m3[index]=m1[index]+m2[index];
}

// function minus
// m1, m2 are vectors, matrices or tensors of the same size
// 1 block, 1 dimensional block size
__global__ void minus(float* m1, float* m2, float* m3, int n){
    for(int index=threadIdx.x;index<n;index+=blockDim.x)
        m3[index]=m1[index]-m2[index];
}

// function mul
// 1 block, 1 dimensional block size
__global__ void mul(float* m1, float factor, float* m2, int n){
    for(int index=threadIdx.x;index<n;index+=blockDim.x)
        m2[index]=m1[index]*factor;
}

// function div
// 1 block, 1 dimensional block size
__global__ void div(float* m1, float divsor, float* m2, int n){
    for(int index=threadIdx.x;index<n;index+=blockDim.x)
        m2[index]=m1[index]/divsor;
}

// function pow
// 1 block, 1 dimensional block size
__global__ void pow(float* m1, float power, float* m2, int n){
    for(int index=threadIdx.x;index<n;index+=blockDim.x)
        m2[index]=pow(m1[index],power);
}

// function sqrt
// 1 block, 1 dimensional block size
__global__ void sqrt(float* m1, float* m2, int n){
    for(int index=threadIdx.x;index<n;index+=blockDim.x)
        m2[index]=sqrt(m1[index]);
}

// function dot
// m1 of shape H*L, m2 of shape L*M and m3 of shape H*M
// 1 block, 2 dimensional block size
__global__ void dot(float* m1, float* m2, float* m3,
    int size_h, int size_l, int size_m){
    int h=size_h, l=size_l, m=size_m;

    for(int h_index=threadIdx.x;h_index<h;h_index+=blockDim.x){
        for(int m_index=threadIdx.y;m_index<m;m_index+=blockDim.y){
            float value=0.0;
            for(int l_index=0;l_index<l;l_index++){
                float x=m1[h_index*l+l_index];
                float y=m2[l_index*m+m_index];
                value+=x*y;
            }
            m3[h_index*m+m_index]=value;
        }
    }
}

